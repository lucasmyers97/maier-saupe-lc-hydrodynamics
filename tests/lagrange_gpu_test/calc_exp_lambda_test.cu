#include "hip/hip_runtime.h"
#include <boost/test/tools/interface.hpp>
#define BOOST_TEST_DYN_LINK
#include <boost/test/unit_test.hpp>
#include "sphere_lebedev_rule.hpp"
#include <cmath>
#define private public
#include "LagrangeMultiplierGPU.hpp"

namespace utf = boost::unit_test;

namespace{
    constexpr int order{590};
    constexpr int vec_dim{5};
    constexpr int space_dim{3};
    constexpr int n_lambda_vals{32}; // 2^vec_dim
}


__global__
void calcExpLambdaTest
(const double *lebedev_coords, const double *lebedev_weights,
 const double *lambda_vals, double *exp_lambda_vals)
{
    extern __shared__ LagrangeMultiplierGPU<double, order, vec_dim> lm[];
    const int thread_idx = threadIdx.x;
    const int n_threads = blockDim.x;

    // parse shared pointer so very end corresponds to shared lebedev data
    int n_lagrange_multipliers = n_threads;
    double *s_lebedev_coords = (double*)&lm[n_lagrange_multipliers];
    double *s_lebedev_weights = (double*)&s_lebedev_coords[space_dim*order];

    // read lebedev data from global memory into shared memory
    lm[thread_idx].readLebedevGlobal(lebedev_coords, lebedev_weights,
                                     thread_idx, n_threads,
                                     s_lebedev_coords, s_lebedev_weights);

    // make each of the LagrangeMultiplierGPU instances point to shared
    // lebedev data
    lm[thread_idx].setLebedevData(s_lebedev_coords, s_lebedev_weights);

    int row_idx{};
    for (int i = 0; i < n_lambda_vals; ++i)
    {
        for (int j = 0; j < vec_dim; ++j)
            lm[thread_idx].Lambda[j] = lambda_vals[vec_dim*i + j];

        for (int j = 0; j < order; ++j)
        {
            row_idx = space_dim*j;
            exp_lambda_vals[order*i + j] 
                = lm[thread_idx].calcExpLambda(row_idx);
        }
    }
}

BOOST_AUTO_TEST_CASE(calc_exp_lambda_test, *utf::tolerance(1e-12))
{
    // get initial copy of lebedev data
    double *x, *y, *z, *w;
    x = new double[order];
    y = new double[order];
    z = new double[order];
    w = new double[order];
    ld_by_order(order, x, y, z, w);

    // make n_blocks copies of lebedev data
    double *lebedev_coords;
    double *lebedev_weights;
    lebedev_coords = new double[space_dim*order];
    lebedev_weights = new double[order];

    for (int i = 0; i < order; ++i)
    {
        lebedev_coords[space_dim*i] = x[i];
        lebedev_coords[space_dim*i + 1] = y[i];
        lebedev_coords[space_dim*i + 2] = z[i];
        lebedev_weights[i] = w[i];
    }

    double *d_lebedev_coords, *d_lebedev_weights;
    hipMalloc(&d_lebedev_coords, space_dim*order*sizeof(double));
    hipMalloc(&d_lebedev_weights, order*sizeof(double));

    hipMemcpy(d_lebedev_coords, lebedev_coords, 
               space_dim*order*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lebedev_weights, lebedev_weights,
               order*sizeof(double), hipMemcpyHostToDevice);

    delete[] lebedev_coords;
    delete[] lebedev_weights;

    double *lambda_vals = new double[n_lambda_vals*vec_dim];
    
    for (int i = 0; i < n_lambda_vals; ++i)
    {
        std::bitset<vec_dim> lambda_bits(i);
        for (int j = 0; j < vec_dim; ++j)
            lambda_vals[vec_dim*i + j] = lambda_bits[j];
    }

    double *d_lambda_vals;
    hipMalloc(&d_lambda_vals, n_lambda_vals*vec_dim*sizeof(double));
    hipMemcpy(d_lambda_vals, lambda_vals, 
               n_lambda_vals*vec_dim*sizeof(double), hipMemcpyHostToDevice);

    double *d_exp_lambda_vals;
    hipMalloc(&d_exp_lambda_vals, n_lambda_vals*order*sizeof(double));
    
    size_t s_mem_size = sizeof(LagrangeMultiplierGPU<double, order, vec_dim>)
                        + space_dim*order*sizeof(double)
                        + order*sizeof(double);
    calcExpLambdaTest<<<1, 1, s_mem_size>>>
        (d_lebedev_coords, d_lebedev_weights, d_lambda_vals, d_exp_lambda_vals);

    double *exp_lambda_vals = new double[n_lambda_vals*order];
    hipMemcpy(exp_lambda_vals, d_exp_lambda_vals, 
               n_lambda_vals*order*sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < n_lambda_vals; ++i)
    {
        for (int j = 0; j < order; ++j)
        {
            
        }
    }
}