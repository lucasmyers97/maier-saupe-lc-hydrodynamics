#include "hip/hip_runtime.h"
#include <boost/test/tools/interface.hpp>
#define BOOST_TEST_DYN_LINK
#include <boost/test/unit_test.hpp>
#include "sphere_lebedev_rule.hpp"
#define private public
#include "LagrangeMultiplierGPU.hpp"

namespace{
    constexpr int order{590};
    constexpr int vec_dim{5};
    constexpr int space_dim{3};
}

__global__
void readLebedevGlobal(double *lebedev_coords, double *lebedev_weights, int n_threads,
                       double *out_lebedev_coords, double *out_lebedev_weights)
{
    extern __shared__ LagrangeMultiplierGPU<double, order, vec_dim> lm[];

    int n_lagrange_multipliers = n_threads;
    double *s_lebedev_coords = (double*)&lm[n_lagrange_multipliers];
    double *s_lebedev_weights = (double*)&s_lebedev_coords[space_dim*order];

    lm[0].readLebedevGlobal(lebedev_coords, lebedev_weights,
                            s_lebedev_coords, s_lebedev_weights,
                            threadIdx.x, n_threads);
    
    int x_coord{};
    int y_coord{};
    int z_coord{};
    for (int i = 0; i < order; ++i)
    {
        x_coord = i*space_dim;
        y_coord = i*space_dim + 1;
        z_coord = i*space_dim + 2;

        out_lebedev_coords[x_coord] = s_lebedev_coords[x_coord];
        out_lebedev_coords[y_coord] = s_lebedev_coords[y_coord];
        out_lebedev_coords[z_coord] = s_lebedev_coords[z_coord];
        out_lebedev_weights[i] = s_lebedev_weights[i];
    }
}

BOOST_AUTO_TEST_CASE(set_lebedev_test)
{
    double *x, *y, *z, *w;
    x = new double[order];
    y = new double[order];
    z = new double[order];
    w = new double[order];

    ld_by_order(order, x, y, z, w);

    double *lebedev_coords = new double[3*order];
    double *lebedev_weights = new double[order];

    for (int i = 0; i < order; ++i)
    {
        lebedev_coords[space_dim*i] = x[i];
        lebedev_coords[space_dim*i + 1] = y[i];
        lebedev_coords[space_dim*i + 2] = z[i];
        lebedev_weights[i] = w[i];
    }

    double *d_lebedev_coords;
    double *d_lebedev_weights;
    hipMalloc(&d_lebedev_coords, space_dim*order*sizeof(double));
    hipMalloc(&d_lebedev_weights, order*sizeof(double));

    hipMemcpy(d_lebedev_coords, lebedev_coords,
               space_dim*order*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lebedev_weights, lebedev_weights,
               order*sizeof(double), hipMemcpyHostToDevice);

    for (int i = 0; i < order; ++i)
    {
        lebedev_coords[space_dim*i] = 0;
        lebedev_coords[space_dim*i + 1] = 0;
        lebedev_coords[space_dim*i + 2] = 0;
        lebedev_weights[i] = 0;
    }

    double *d_out_lebedev_coords;
    double *d_out_lebedev_weights;
    hipMalloc(&d_out_lebedev_coords, space_dim*order*sizeof(double));
    hipMalloc(&d_out_lebedev_weights, order*sizeof(double));

    unsigned long shared_mem_size 
        = sizeof(LagrangeMultiplierGPU<double, order, vec_dim>)
          + space_dim*order*sizeof(double)
          + order*sizeof(double);

    int n_threads = 1;
    readLebedevGlobal <<<1, n_threads, shared_mem_size>>>
		(d_lebedev_coords, d_lebedev_weights, n_threads,
        d_out_lebedev_coords, d_out_lebedev_weights);

    hipError_t error = hipPeekAtLastError();
    BOOST_TEST_REQUIRE(error == 0, "cuda had errors");

    hipFree(d_lebedev_coords);
    hipFree(d_lebedev_weights);

    hipMemcpy(lebedev_coords, d_out_lebedev_coords, 
               space_dim*order*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(lebedev_weights, d_out_lebedev_weights, 
               order*sizeof(double), hipMemcpyDeviceToHost);

    error = hipPeekAtLastError();
    BOOST_TEST_REQUIRE(error == 0, "cuda had errors");

    hipFree(d_out_lebedev_coords);
    hipFree(d_out_lebedev_weights);
            
    for (int i = 0; i < order; ++i)
    {
        BOOST_TEST(lebedev_coords[space_dim*i] == x[i]);
        BOOST_TEST(lebedev_coords[space_dim*i + 1] == y[i]);
        BOOST_TEST(lebedev_coords[space_dim*i + 2] == z[i]);
        BOOST_TEST(lebedev_weights[i] == w[i]);
    }

    delete[] lebedev_coords;
    delete[] lebedev_weights;
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] w;
}