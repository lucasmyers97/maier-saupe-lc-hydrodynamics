#include "hip/hip_runtime.h"
#include <boost/test/tools/interface.hpp>
#define BOOST_TEST_DYN_LINK
#include <boost/test/unit_test.hpp>
#include "sphere_lebedev_rule.hpp"
#include <iostream>
#define private public
#include "LagrangeMultiplierGPU.hpp"

namespace utf = boost::unit_test;

namespace{
    constexpr int order{590};
    constexpr int vec_dim{5};
    constexpr int space_dim{3};
    constexpr int max_iters{30};
    constexpr double tol{1e-12};
}

__global__
void calcLambdaTest
(const double *lebedev_coords, const double *lebedev_weights, double *Q)
{
    extern __shared__ LagrangeMultiplierGPU<double, order, vec_dim> lm[];
    const int thread_idx = threadIdx.x;
    const int n_threads = blockDim.x;

    // parse shared pointer so very end corresponds to shared lebedev data
    int n_lagrange_multipliers = n_threads;
    double *s_lebedev_coords = (double*)&lm[n_lagrange_multipliers];
    double *s_lebedev_weights = (double*)&s_lebedev_coords[space_dim*order];

    if (thread_idx == 0)
    {
        // read lebedev data from global memory into shared memory
        lm[thread_idx].readLebedevGlobal(lebedev_coords, lebedev_weights,
                                        thread_idx, n_threads,
                                        s_lebedev_coords, s_lebedev_weights);

        // make each of the LagrangeMultiplierGPU instances point to shared
        // lebedev data
        lm[thread_idx].setLebedevData(s_lebedev_coords, s_lebedev_weights);

        lm[thread_idx].setParams(tol, max_iters);
        lm[thread_idx].calcLambda(Q);
    }
}

BOOST_AUTO_TEST_CASE(calc_lambda_test)
{
    // get initial copy of lebedev data
    double *x, *y, *z, *w;
    x = new double[order];
    y = new double[order];
    z = new double[order];
    w = new double[order];
    ld_by_order(order, x, y, z, w);

    // make n_blocks copies of lebedev data
    double *lebedev_coords;
    double *lebedev_weights;
    lebedev_coords = new double[space_dim*order];
    lebedev_weights = new double[order];

    for (int i = 0; i < order; ++i)
    {
        lebedev_coords[space_dim*i] = x[i];
        lebedev_coords[space_dim*i + 1] = y[i];
        lebedev_coords[space_dim*i + 2] = z[i];
        lebedev_weights[i] = w[i];
    }

    delete[] x;
    delete[] y;
    delete[] z;
    delete[] w;

    double *d_lebedev_coords, *d_lebedev_weights;
    hipMalloc(&d_lebedev_coords, space_dim*order*sizeof(double));
    hipMalloc(&d_lebedev_weights, order*sizeof(double));

    hipMemcpy(d_lebedev_coords, lebedev_coords, 
               space_dim*order*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lebedev_weights, lebedev_weights,
               order*sizeof(double), hipMemcpyHostToDevice);

    delete[] lebedev_coords;
    delete[] lebedev_weights;

    double *Q = new double[max_iters*vec_dim];
    Q[0] = 0.6;
    Q[1] = 0;
    Q[2] = 0;
    Q[3] = -0.3;
    Q[4] = 0;

    double *d_Q;
    hipMalloc(&d_Q, max_iters*vec_dim*sizeof(double));
    hipMemcpy(d_Q, Q, max_iters*vec_dim*sizeof(double), hipMemcpyHostToDevice);

    size_t s_mem_size = sizeof(LagrangeMultiplierGPU<double, order, vec_dim>)
                        + space_dim*order*sizeof(double)
                        + order*sizeof(double);
    calcLambdaTest <<<1, 1, s_mem_size>>>
        (d_lebedev_coords, d_lebedev_weights, d_Q);

    delete[] Q;
    hipFree(d_Q);

    hipError_t error = hipPeekAtLastError();
    BOOST_TEST(error == 0);
}