
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

int main()
{
    hipDeviceProp_t *prop = new hipDeviceProp_t;
    int *device = new int;
    hipError_t error = hipGetDevice(device);
    assert(error == 0);
    error = hipGetDeviceProperties(prop, *device);
    assert(error == 0);

    std::cout << "Total global memory: " 
              << prop->totalGlobalMem << std::endl;
    std::cout << "Maximum threads per multiprocessor: "
              << prop->maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Maximum threads per block: "
              << prop->maxThreadsPerBlock << std::endl;
    std::cout << "Shared memory per multiprocessor: "
              << prop->sharedMemPerMultiprocessor << std::endl;
    std::cout << "Shared memory per block: "
              << prop->sharedMemPerBlock << std::endl;
    std::cout << "Number of multiprocessors: "
              << prop->multiProcessorCount << std::endl;

    std::cout << "Max dynamic shared memory size: "
              << hipFuncAttributeMaxDynamicSharedMemorySize << std::endl;
    std::cout << "Preferred shared memory carveout: "
              << hipFuncAttributePreferredSharedMemoryCarveout << std::endl;

    return 0;
}